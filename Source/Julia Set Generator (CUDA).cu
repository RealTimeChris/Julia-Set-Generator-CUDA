#include "hip/hip_runtime.h"
// Julia Set Generator (CUDA).cu
// July 2018 (Updated September 2019)
// Generates and saves Julia Sets, with custom color gradients.
// Chris M
// https://github.com/RealTimeChris


#if !defined WIN32_LEAN_AND_MEAN
#define WIN32_LEAN_AND_MEAN
#endif

#include <Windows.h>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"


// Resolution of the output image.
#define HEIGHT 4320
#define WIDTH 7680


// Collects the real and imaginary bounds of a section of the complex plane.
void planeCollect(double *realLeft, double *realRight, double *imagUpper, double *imagLower)
{
	char realL[256], realR[256], imagU[256], imagL[256];

	printf("Enter the left bound of the complex plane: "); gets_s(realL);
	printf("Enter the right bound of the complex plane: "); gets_s(realR);
	printf("Enter the upper bound of the complex plane: "); gets_s(imagU);
	printf("Enter the lower bound of the complex plane: "); gets_s(imagL);

	sscanf_s(realL, "%lf", realLeft);
	sscanf_s(realR, "%lf", realRight);
	sscanf_s(imagU, "%lf", imagUpper);
	sscanf_s(imagL, "%lf", imagLower);

	return;
}


// Calculates the real and imaginary delta per pixel along each axis.
void intervalCalc(double realLeft, double realRight, double imagUpper,double imagLower, double* realYInt, double* imagXInt)
{
	*realYInt = (realRight - realLeft) / (WIDTH - 1);
	*imagXInt = (imagUpper - imagLower) / (HEIGHT - 1);

	return;
}


// Collects the c value for use in: z_n = z^2 + c
void complexGet(hipDoubleComplex* c)
{
	char aRaw[256], bRaw[256];
	double a, b;

	printf("\nEnter the real part of c: "); gets_s(aRaw);
	printf("Enter the imaginary part of c: "); gets_s(bRaw);

	sscanf_s(aRaw, "%lf", &a);
	sscanf_s(bRaw, "%lf", &b);

	*c = make_hipDoubleComplex(a, b);

	return;
}


// Defines the x and y coordinates (i,j) of the current pixel that is being processed, using the current block and thread values.
// This is where 2D of 2D indexing is abstracted into global 2D indexing.
__device__ void pixelIndex(int* i, int* j)
{
	*i = (blockIdx.x * blockDim.x) + threadIdx.x;
	*j = (blockIdx.y * blockDim.y) + threadIdx.y;

	return;
}


// Calculates where in the row-major linearized array to store the current value.
// This is where the global 2D indexing is abstracted into global 1D indexing.
__device__ void matrixIndex(int i, int j, int *matrixLoc)
{
	*matrixLoc = (i * WIDTH) + j;

	return;
}


// Calculates the real-interval for the current pixel.
__device__ void realCV(double realLeftDev, double realYIntDev, int j, double *currentVal)
{
	*currentVal = realLeftDev + (realYIntDev * j);

	return;
}


// Calculates the imaginary-interval value for the current pixel.
__device__ void imagCV(double imagUpperDev, double imagXIntDev, int i, double *currentVal)
{
	*currentVal = imagUpperDev - (imagXIntDev * i);

	return;
}


// Fills the elements of a matrix that represents the complex plane.
__global__ void complexFill(
	double *realLeftDev, double *realYIntDev, double *imagUpperDev,
	double *imagXIntDev, hipDoubleComplex *complexMatrixDev)
{
	int i, j, matrixLoc;
	double realCurrentVal, imagCurrentVal;
	hipDoubleComplex complexCurrentVal;

	pixelIndex(&i, &j);

	realCV(*realLeftDev, *realYIntDev, j, &realCurrentVal);
	imagCV(*imagUpperDev, *imagXIntDev, i, &imagCurrentVal);

	complexCurrentVal = make_hipDoubleComplex(realCurrentVal, imagCurrentVal);

	matrixIndex(i, j, &matrixLoc);

	complexMatrixDev[matrixLoc] = complexCurrentVal;

	return;
}


// Tracks the number of iterations each pixel takes to diverge.
__device__ void divCheck(hipDoubleComplex zN, int *divCountDev, int matrixLoc)
{
	int notDiverged;

	notDiverged = hipCabs(zN) < 2;

	divCountDev[matrixLoc] = (divCountDev[matrixLoc]) + notDiverged;

	return;
}


// Fills the elements of a matrix with the results of z_n = z^2 + c, and executes divergence-counting.
__global__ void zNCalc(hipDoubleComplex *zMatrix, hipDoubleComplex *c, int *divCountDev)
{

	for (int iter = 0; iter < 100; iter++)
	{
		int i, j, matrixLoc;
		hipDoubleComplex zN, z;

		pixelIndex(&i, &j);

		matrixIndex(i, j, &matrixLoc);

		z = zMatrix[matrixLoc];

		zN = hipCadd(hipCmul(z, z), *c);

		divCheck(zN, divCountDev, matrixLoc);

		zMatrix[matrixLoc] = zN;
	}

	return;
}


// Collects the a,k,c, and d transform values for each color layer.
void gradientCollect(
	float *redD, float *redK, float *redA, float *redC,
	float *greenD, float *greenK, float *greenA, float *greenC,
	float *blueD, float *blueK, float *blueA, float *blueC)
{
	char redDS[256], redKS[256], redAS[256], redCS[256];
	char greenDS[256], greenKS[256], greenAS[256], greenCS[256];
	char blueDS[256], blueKS[256], blueAS[256], blueCS[256];

	printf("COLOR GRADIENT DESIGN:\n");

	printf("Enter d (red): "); gets_s(redDS);
	printf("Enter k (red): "); gets_s(redKS);
	printf("Enter a (red): "); gets_s(redAS);
	printf("Enter c (red): "); gets_s(redCS);

	printf("\nEnter d (green): "); gets_s(greenDS);
	printf("Enter k (green): "); gets_s(greenKS);
	printf("Enter a (green): "); gets_s(greenAS);
	printf("Enter c (green): "); gets_s(greenCS);

	printf("\nEnter d (blue): "); gets_s(blueDS);
	printf("Enter k (blue): "); gets_s(blueKS);
	printf("Enter a (blue): "); gets_s(blueAS);
	printf("Enter c (blue): "); gets_s(blueCS);

	sscanf_s(redDS, "%f", redD);
	sscanf_s(redKS, "%f", redK);
	sscanf_s(redAS, "%f", redA);
	sscanf_s(redCS, "%f", redC);

	sscanf_s(greenDS, "%f", greenD);
	sscanf_s(greenKS, "%f", greenK);
	sscanf_s(greenAS, "%f", greenA);
	sscanf_s(greenCS, "%f", greenC);

	sscanf_s(blueDS, "%f", blueD);
	sscanf_s(blueKS, "%f", blueK);
	sscanf_s(blueAS, "%f", blueA);
	sscanf_s(blueCS, "%f", blueC);

	return;
}


// Transforms the divergence-count matrix into a single color layer for the RGB matrix output.
__global__ void colorTransform(
	int *divCountDev, unsigned char *colorLayerDev, float *colorDDev,
	float *colorKDev, float *colorADev, float *colorCDev)
{
	int i, j, matrixLoc;
	float colorValue;

	pixelIndex(&i, &j);

	matrixIndex(i, j, &matrixLoc);

	colorValue = divCountDev[matrixLoc];

	if (colorValue == 100)
	{
		colorValue = 0;
	}
	else
	{
		colorValue = colorValue - (*colorDDev);
		colorValue = (*colorKDev) * colorValue;
		colorValue = sinf(colorValue);
		colorValue = (*colorADev) * colorValue;
		colorValue = colorValue + (*colorCDev);
		colorValue = roundf(colorValue);

		if (colorValue > 255)
		{
			colorValue = 255;
		}
		else if (colorValue < 0)
		{
			colorValue = 0;
		}
	}

	colorLayerDev[matrixLoc] = (unsigned char)colorValue;

	return;
}


// Saves the row-major linearized RGB arrays as a bitmap image on the Windows Desktop.
void bmpSave(unsigned char *redLayer, unsigned char *greenLayer, unsigned char *blueLayer)
{
	char *pathUser;
	pathUser = (char *)malloc(strlen(getenv("USERPROFILE")));
	pathUser = { getenv("USERPROFILE") };

	char pathDesktop[12] = { "\\Desktop\\" };

	char pathFName[256];
	printf("Enter a file name: "); gets_s(pathFName);

	char pathFExt[5] = { ".bmp" };

	char *filePath;
	filePath = (char *)malloc(strlen(getenv("USERPROFILE")) + strlen(pathDesktop) + strlen(pathFName) + strlen(pathFExt) + 1);

	strcpy(filePath, pathUser);
	strcat(filePath, pathDesktop);
	strcat(filePath, pathFName);
	strcat(filePath, pathFExt);

	unsigned char bmpFileHeader[14] = { 'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0 };
	unsigned char bmpFileInfoHeader[40] = { 40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0 };

	int fileSize = (54 + (3 * HEIGHT * WIDTH));

	bmpFileHeader[2] = (unsigned char)(fileSize);
	bmpFileHeader[3] = (unsigned char)(fileSize >> 8);
	bmpFileHeader[4] = (unsigned char)(fileSize >> 16);
	bmpFileHeader[5] = (unsigned char)(fileSize >> 24);

	bmpFileInfoHeader[4] = (unsigned char)(WIDTH);
	bmpFileInfoHeader[5] = (unsigned char)(WIDTH >> 8);
	bmpFileInfoHeader[6] = (unsigned char)(WIDTH >> 16);
	bmpFileInfoHeader[7] = (unsigned char)(WIDTH >> 24);
	bmpFileInfoHeader[8] = (unsigned char)(HEIGHT);
	bmpFileInfoHeader[9] = (unsigned char)(HEIGHT >> 8);
	bmpFileInfoHeader[10] = (unsigned char)(HEIGHT >> 16);
	bmpFileInfoHeader[11] = (unsigned char)(HEIGHT >> 24);

	unsigned char *bmpFileComplete;
	bmpFileComplete = (unsigned char *)malloc(fileSize * sizeof(unsigned char));

	for (int i = 0; i < 14; i++) bmpFileComplete[i] = bmpFileHeader[i];
	for (int i = 0; i < 40; i++) bmpFileComplete[14 + i] = bmpFileInfoHeader[i];

	// Flip the color layers along the x-axis of each matrix in accordance with the bitmap file format.
	unsigned char *redLayerFlip, *greenLayerFlip, *blueLayerFlip;
	redLayerFlip = (unsigned char *)malloc((HEIGHT * WIDTH) * sizeof(unsigned char));
	greenLayerFlip = (unsigned char *)malloc((HEIGHT * WIDTH) * sizeof(unsigned char));
	blueLayerFlip = (unsigned char *)malloc((HEIGHT * WIDTH) * sizeof(unsigned char));

	for (int i = 0; i < HEIGHT; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			redLayerFlip[(HEIGHT - 1 - i) * WIDTH + j] = redLayer[i * WIDTH + j];
			greenLayerFlip[(HEIGHT - 1 - i) * WIDTH + j] = greenLayer[i * WIDTH + j];
			blueLayerFlip[(HEIGHT - 1 - i) * WIDTH + j] = blueLayer[i * WIDTH + j];
		}
	}

	printf("\nSaving the image to desktop...");
	for (int i = 0; i < (3 * HEIGHT * WIDTH); i += 3)
	{
		bmpFileComplete[(54 + i) + 2] = redLayerFlip[i / 3];
		bmpFileComplete[(54 + i) + 1] = greenLayerFlip[i / 3];
		bmpFileComplete[(54 + i) + 0] = blueLayerFlip[i / 3];
	}

	FILE *file;
	file = fopen(filePath, "wb+");

	fwrite(bmpFileComplete, sizeof(unsigned char), (54 + (3 * HEIGHT * WIDTH)), file);

	fclose(file);
	printf(" Done!\n\n");

	return;
}


// Main loop for the console program.
int main(void)
{
	// Welcome message.
	printf("Julia Set Generator (CUDA)\nCreated by Chris M\nhttps://github.com/RealTimeChris\n\n");
	printf("WARNING: There is no data validation or exception handling, so watch your input to avoid undefined behavior.\n\n");

	while (1 == 1)
	{
		// Grid and block values.
		dim3 threadsPB(8, 128);
		dim3 grid(HEIGHT / threadsPB.x, WIDTH / threadsPB.y);

		// Host copies of user input.
		double realLeft, realRight, realYInt, imagUpper, imagLower, imagXInt;
		hipDoubleComplex c;

		// Collect and prepare user input.
		printf("JULIA SET PARAMETERS:\n");
		planeCollect(&realLeft, &realRight, &imagUpper, &imagLower);
		complexGet(&c);
		intervalCalc(realLeft, realRight, imagUpper, imagLower, &realYInt, &imagXInt);

		// Device copies of the necessary values.
		printf("\nGetting the GPU warmed up...");
		double* realLeftDev;
		hipMalloc((void **)&realLeftDev, sizeof(double));
		hipMemcpy(realLeftDev, &realLeft, sizeof(double), hipMemcpyHostToDevice);

		double* realYIntDev;
		hipMalloc((void **)&realYIntDev, sizeof(double));
		hipMemcpy(realYIntDev, &realYInt, sizeof(double), hipMemcpyHostToDevice);

		double* imagUpperDev;
		hipMalloc((void **)&imagUpperDev, sizeof(double));
		hipMemcpy(imagUpperDev, &imagUpper, sizeof(double), hipMemcpyHostToDevice);

		double* imagXIntDev;
		hipMalloc((void **)&imagXIntDev, sizeof(double));
		hipMemcpy(imagXIntDev, &imagXInt, sizeof(double), hipMemcpyHostToDevice);

		hipDoubleComplex* cDev;
		hipMalloc((void **)&cDev, sizeof(hipDoubleComplex));
		hipMemcpy(cDev, &c, sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

		// Device copy of the complex matrix.
		hipDoubleComplex *complexMatrixDev;
		hipMalloc((void **)&complexMatrixDev, (HEIGHT * WIDTH) * sizeof(hipDoubleComplex));
		printf(" Done!\n\n");

		// Fill the complex matrix on the device.
		printf("Creating the complex plane...");
		complexFill << <grid, threadsPB >> > (realLeftDev, realYIntDev, imagUpperDev, imagXIntDev, complexMatrixDev);
		hipDeviceSynchronize();
		printf(" Done!\n\n");

		// Free up some device memory.
		hipFree(realLeftDev);
		hipFree(realYIntDev);
		hipFree(imagUpperDev);
		hipFree(imagXIntDev);

		// Device copy of divergence-count matrix.
		int *divCountDev;
		hipMalloc((void **)&divCountDev, (HEIGHT * WIDTH) * sizeof(int));

		// Fill the divergence-count matrix by executing the iterations of the function.
		printf("Executing 100 iterations of 33.18 million instances of z_n = z^2 + c...");
		zNCalc << <grid, threadsPB >> > (complexMatrixDev, cDev, divCountDev);
		hipDeviceSynchronize();
		printf(" Done!\n\n");

		// Free up some device memory.
		hipFree(complexMatrixDev);
		hipFree(cDev);

		// Host copies of variables for the color gradient.
		float redD, redK, redA, redC;
		float greenD, greenK, greenA, greenC;
		float blueD, blueK, blueA, blueC;

		// Collect the transform values for creating the color gradient.
		gradientCollect(
			&redD, &redK, &redA, &redC,
			&greenD, &greenK, &greenA, &greenC,
			&blueD, &blueK, &blueA, &blueC);

		// Device copies of the color gradient transform values.
		float *redDDev, *redKDev, *redADev, *redCDev;
		float *greenDDev, *greenKDev, *greenADev, *greenCDev;
		float *blueDDev, *blueKDev, *blueADev, *blueCDev;

		hipMalloc((void **)&redDDev, sizeof(float));
		hipMemcpy(redDDev, &redD, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&redKDev, sizeof(float));
		hipMemcpy(redKDev, &redK, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&redADev, sizeof(float));
		hipMemcpy(redADev, &redA, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&redCDev, sizeof(float));
		hipMemcpy(redCDev, &redC, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&greenDDev, sizeof(float));
		hipMemcpy(greenDDev, &greenD, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&greenKDev, sizeof(float));
		hipMemcpy(greenKDev, &greenK, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&greenADev, sizeof(float));
		hipMemcpy(greenADev, &greenA, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&greenCDev, sizeof(float));
		hipMemcpy(greenCDev, &greenC, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&blueDDev, sizeof(float));
		hipMemcpy(blueDDev, &blueD, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&blueKDev, sizeof(float));
		hipMemcpy(blueKDev, &blueK, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&blueADev, sizeof(float));
		hipMemcpy(blueADev, &blueA, sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void **)&blueCDev, sizeof(float));
		hipMemcpy(blueCDev, &blueC, sizeof(float), hipMemcpyHostToDevice);

		// Device copy of the red layer.
		unsigned char *redLayerDev;
		hipMalloc((void **)&redLayerDev, (HEIGHT * WIDTH) * sizeof(unsigned char));

		// Create the red layer.
		printf("\nTransforming the red layer...");
		colorTransform << <grid, threadsPB >> > (divCountDev, redLayerDev, redDDev, redKDev, redADev, redCDev);
		hipDeviceSynchronize();
		printf(" Done!\n");

		// Host copy of the red layer.
		unsigned char *redLayer;
		redLayer = (unsigned char *)malloc((HEIGHT * WIDTH) * sizeof(unsigned char));
		hipMemcpy(redLayer, redLayerDev, (HEIGHT * WIDTH) * sizeof(unsigned char), hipMemcpyDeviceToHost);

		// Free up some device memory.
		hipFree(redLayerDev);
		hipFree(redDDev);
		hipFree(redKDev);
		hipFree(redADev);
		hipFree(redCDev);

		// Device copy of the green layer.
		unsigned char *greenLayerDev;
		hipMalloc((void **)&greenLayerDev, (HEIGHT * WIDTH) * sizeof(unsigned char));

		// Create the green layer.
		printf("Transforming the green layer...");
		colorTransform << <grid, threadsPB >> > (divCountDev, greenLayerDev, greenDDev, greenKDev, greenADev, greenCDev);
		hipDeviceSynchronize();
		printf(" Done!\n");

		// Host copy of the green layer.
		unsigned char *greenLayer;
		greenLayer = (unsigned char *)malloc((HEIGHT * WIDTH) * sizeof(unsigned char));
		hipMemcpy(greenLayer, greenLayerDev, (HEIGHT * WIDTH) * sizeof(unsigned char), hipMemcpyDeviceToHost);

		// Free up some device memory.
		hipFree(greenLayerDev);
		hipFree(greenDDev);
		hipFree(greenKDev);
		hipFree(greenADev);
		hipFree(greenCDev);

		// Device copy of the blue layer.
		unsigned char *blueLayerDev;
		hipMalloc((void **)&blueLayerDev, (HEIGHT * WIDTH) * sizeof(unsigned char));

		// Create the blue layer.
		printf("Transforming the blue layer...");
		colorTransform << <grid, threadsPB >> > (divCountDev, blueLayerDev, blueDDev, blueKDev, blueADev, blueCDev);
		hipDeviceSynchronize();
		printf(" Done!\n\n");

		// Host copy of the blue layer.
		unsigned char *blueLayer;
		blueLayer = (unsigned char *)malloc((HEIGHT * WIDTH) * sizeof(unsigned char));
		hipMemcpy(blueLayer, blueLayerDev, (HEIGHT * WIDTH) * sizeof(unsigned char), hipMemcpyDeviceToHost);

		// Free up some device memory.
		hipFree(blueLayerDev);
		hipFree(blueDDev);
		hipFree(blueKDev);
		hipFree(blueADev);
		hipFree(blueCDev);
		hipFree(divCountDev);

		// Render the image to disk.
		bmpSave(redLayer, greenLayer, blueLayer);

		// Free up some host memory.
		free(redLayer);
		free(greenLayer);
		free(blueLayer);

		// Ask for another run.
		char progRepeat[12];
		printf("Would you like to create another? (y/n) "); gets_s(progRepeat);
		printf("\n");

		if (progRepeat[0] != 'y')
		{
			break;
		}
	}

	char progEnd[256];
	printf("Press enter to exit...");
	gets_s(progEnd);

	return 0;
}
